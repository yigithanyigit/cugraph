#include <cugraph/algorithms.hpp>
#include <cugraph/graph.hpp>
#include <cugraph/detail/shuffle_wrappers.hpp>
#include <cugraph/detail/utility_wrappers.hpp>
#include <cugraph/graph_functions.hpp>
#include <cugraph/graph_generators.hpp>
#include <cugraph/legacy/functions.hpp>  // legacy coo_to_csr


#include <raft/core/handle.hpp>
#include <rmm/device_uvector.hpp>
#include <iostream>

#include "mmio.h"

void mtx_parser(const std::string& file_path, std::vector<int32_t>& h_src, std::vector<int32_t>& h_dst, std::vector<float>& h_weights) {
    FILE* f = fopen(file_path.c_str(), "r");
    if (f == NULL) {
        std::cerr << "Error opening file: " << file_path << std::endl;
        exit(1);
    }

    MM_typecode matcode;
    if (mm_read_banner(f, &matcode) != 0) {
        std::cerr << "Error reading matrix banner" << std::endl;
        exit(1);
    }

    if (!mm_is_matrix(matcode) || !mm_is_sparse(matcode) || !mm_is_real(matcode)) {
        std::cerr << "This example only works with real-valued sparse matrices" << std::endl;
        exit(1);
    }

    int M, N, nz;
    if (mm_read_mtx_crd_size(f, &M, &N, &nz) != 0) {
        std::cerr << "Error reading matrix size" << std::endl;
        exit(1);
    }

    h_src.resize(nz);
    h_dst.resize(nz);
    h_weights.resize(nz);

    for (int i = 0; i < nz; i++) {
        if (fscanf(f, "%d %d %f\n", &h_src[i], &h_dst[i], &h_weights[i]) != 3) {
            std::cerr << "Error reading matrix entry" << std::endl;
            exit(1);
        }
        h_src[i]--;
        h_dst[i]--;
    }
    fclose(f);
}


int main(int argc, char** argv) {
    // This is looks necessary to run cugraph algorithms?
    raft::handle_t handle{};

    /*
    // Create example graph data - a simple graph with 5 vertices and 6 edges
    std::vector<int32_t> h_src = {0, 1, 1, 2, 2, 3};
    std::vector<int32_t> h_dst = {1, 2, 3, 3, 4, 4};
    std::vector<float> h_weights = {1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f};
    */

    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <path to mtx file>" << std::endl;
        exit(1);
    } else {
        std::cout << "Reading graph from file: " << argv[1] << std::endl;
    }

    std::string file_path = argv[1];

    std::vector<int32_t> h_src;
    std::vector<int32_t> h_dst;
    std::vector<float> h_weights;

    mtx_parser(file_path, h_src, h_dst, h_weights);

    // Create device vectors
    rmm::device_uvector<int32_t> d_src(h_src.size(), handle.get_stream());
    rmm::device_uvector<int32_t> d_dst(h_dst.size(), handle.get_stream());
    rmm::device_uvector<float> d_weights(h_weights.size(), handle.get_stream());

    // Copy data to GPU
    hipMemcpy(d_src.data(), h_src.data(), sizeof(int32_t) * h_src.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_dst.data(), h_dst.data(), sizeof(int32_t) * h_dst.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_weights.data(), h_weights.data(), sizeof(float) * h_weights.size(), hipMemcpyHostToDevice);

    // Create graph
    cugraph::graph_t<int32_t, int32_t, false, false> graph(handle);
    std::optional<cugraph::edge_property_t<cugraph::graph_view_t<int32_t, int32_t, false, false>, float>> edge_weights;
    std::optional<rmm::device_uvector<int32_t>> renumber_map{std::nullopt};
    std::tie(graph, edge_weights, std::ignore, std::ignore, renumber_map) = 
        cugraph::create_graph_from_edgelist<int32_t, int32_t, float, int32_t, int32_t, false, false>(
            handle,
            std::nullopt,              // No vertex list needed
            std::move(d_src),
            std::move(d_dst),
            std::make_optional(std::move(d_weights)),
            std::nullopt,              // No edge ids needed
            std::nullopt,              // No edge types needed
            cugraph::graph_properties_t{false, false},  // Not symmetric, allow multi-edges
            false                      // Don't renumber
        );

    // Prepare for Louvain ??? What is clustering?
    rmm::device_uvector<int32_t> clustering(5, handle.get_stream()); // Size = number of vertices

    // Run Louvain
    auto [num_levels, modularity] = cugraph::louvain(
        handle, 
        std::optional<std::reference_wrapper<raft::random::RngState>>{std::nullopt},
        graph.view(),
        edge_weights ? std::make_optional(edge_weights->view()) : std::nullopt,
        clustering.data(),
        100,    // max_level
        1e-7f,  // threshold
        1.0f    // resolution
    );

    // What is clustering?
    std::vector<int32_t> h_clustering(5);
    hipMemcpy(h_clustering.data(), clustering.data(), sizeof(int32_t) * 5, hipMemcpyDeviceToHost);

    std::cout << "Number of levels: " << num_levels << std::endl;
    std::cout << "Modularity: " << modularity << std::endl;
    std::cout << "Communities: ";
    for (auto c : h_clustering) {
        std::cout << c << " ";
    }
    std::cout << std::endl;

    return 0;
}